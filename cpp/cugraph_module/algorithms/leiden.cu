// Copyright (c) 2016-2022 Memgraph Ltd. [https://memgraph.com]
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "mg_cugraph_utility.hpp"

namespace {
// TODO: Check Leiden instances. Update in new cuGraph API.
using vertex_t = int32_t;
using edge_t = int32_t;
using weight_t = double;

constexpr char const *kProcedureLeiden = "get";

constexpr char const *kArgumentMaxIterations = "max_iterations";
constexpr char const *kArgumentResolution = "resolution";

constexpr char const *kResultFieldNode = "node";
constexpr char const *kResultFieldPartition = "partition";

void InsertLeidenRecord(mgp_graph *graph, mgp_result *result, mgp_memory *memory, const std::uint64_t node_id,
                        std::int64_t partition) {
  auto *node = mgp::graph_get_vertex_by_id(graph, mgp_vertex_id{.as_int = static_cast<int64_t>(node_id)}, memory);
  if (!node) {
    if (mgp::graph_is_transactional(graph)) {
      throw mg_exception::InvalidIDException();
    }
    return;
  }

  auto *record = mgp::result_new_record(result);
  if (record == nullptr) throw mg_exception::NotEnoughMemoryException();

  mg_utility::InsertNodeValueResult(record, kResultFieldNode, node, memory);
  mg_utility::InsertIntValueResult(record, kResultFieldPartition, partition, memory);
}

void LeidenProc(mgp_list *args, mgp_graph *graph, mgp_result *result, mgp_memory *memory) {
  try {
    auto max_iterations = mgp::value_get_int(mgp::list_at(args, 0));
    auto resulution = mgp::value_get_double(mgp::list_at(args, 1));

    auto mg_graph = mg_utility::GetGraphView(graph, result, memory, mg_graph::GraphType::kUndirectedGraph);
    if (mg_graph->Empty()) return;

    auto n_vertices = mg_graph.get()->Nodes().size();

    // Define handle and operation stream
    raft::handle_t handle{};
    auto stream = handle.get_stream();

    // IMPORTANT: Leiden cuGraph algorithm works only on legacy code
    auto cu_graph_ptr =
        mg_cugraph::CreateCugraphLegacyFromMemgraph<vertex_t, edge_t, weight_t>(*mg_graph.get(), handle);
    auto cu_graph_view = cu_graph_ptr->view();
    cu_graph_view.prop.directed = false;

    rmm::device_uvector<vertex_t> clustering_result(n_vertices, stream);
    cugraph::leiden<vertex_t, edge_t, weight_t>(handle, cu_graph_view, clustering_result.data(), max_iterations, resulution);

    for (vertex_t node_id = 0; node_id < clustering_result.size(); ++node_id) {
      auto partition = clustering_result.element(node_id, stream);
      InsertLeidenRecord(graph, result, memory, mg_graph->GetMemgraphNodeId(node_id), partition);
    }
  } catch (const std::exception &e) {
    // We must not let any exceptions out of our module.
    mgp::result_set_error_msg(result, e.what());
    return;
  }
}
}  // namespace

extern "C" int mgp_init_module(struct mgp_module *module, struct mgp_memory *memory) {
  mgp_value *default_max_iter;
  mgp_value *default_resolution;
  try {
    auto *leiden_proc = mgp::module_add_read_procedure(module, kProcedureLeiden, LeidenProc);

    default_max_iter = mgp::value_make_int(100, memory);
    default_resolution = mgp::value_make_double(1.0, memory);

    mgp::proc_add_opt_arg(leiden_proc, kArgumentMaxIterations, mgp::type_int(), default_max_iter);
    mgp::proc_add_opt_arg(leiden_proc, kArgumentResolution, mgp::type_float(), default_resolution);

    mgp::proc_add_result(leiden_proc, kResultFieldNode, mgp::type_node());
    mgp::proc_add_result(leiden_proc, kResultFieldPartition, mgp::type_int());
  } catch (const std::exception &e) {
    mgp_value_destroy(default_max_iter);
    mgp_value_destroy(default_resolution);
    return 1;
  }

  mgp_value_destroy(default_max_iter);
  mgp_value_destroy(default_resolution);
  return 0;
}

extern "C" int mgp_shutdown_module() { return 0; }
